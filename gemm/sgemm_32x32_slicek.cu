#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_runtime.h>
#include "gemm_utils.cu"
#include "../utils/buffer.cu"

namespace sgemm_32x32_4x8_SliceK {

/* [WHEN] 48 < K <= 256 */
struct ShapeLayoutSliceK {
    uint32_t brid, bcid, tid, wid, lid;
    uint32_t wrows, wcols, lrid, lcid;
    uint32_t M, N, K, aS, bS, cS;
    // uint32_t slice_len, slice_idx;
    __device__ ShapeLayoutSliceK(const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS) {
        // 线程块与线程的标识
        brid = blockIdx.y; bcid = blockIdx.x; tid = threadIdx.x; wid = tid / 32; lid = tid % 32;
        // 线程束的排列布局
        wrows = 8; wcols = 4;
        lrid = (lid % 16) / 2;
        lcid = (lid / 16) * 2 + (lid % 2);
        // 矩阵形状与跨步
        this->M = M; this->N = N; this->K = K;
        this->aS = aS; this->bS = bS; this->cS = cS;
        // 沿着维度轴 K 切片的标识
        // slice_len = 4; slice_idx = wid;
    }
};

__device__ __forceinline__
void reduce_over_warp(
    float *smem, const uint32_t num_warp, const uint32_t num_datum, const uint32_t wid, const uint32_t lid
) {
    /* 在一个 Block 内的所有 Warp 之上进行归约，假设 num_datum 为 4 的整数倍 */
    for (uint32_t offset = num_warp / 2; offset >= 1; offset /= 2) {
        if (wid < offset) {
            #pragma unroll
            for (uint32_t i = lid * 4; i < num_datum; i += warpSize * 4) {
                float4 my = *reinterpret_cast<float4*>(smem + wid * num_datum + i);
                float4 other = *reinterpret_cast<float4*>(smem + (wid + offset) * num_datum + i);
                my.x += other.x; my.y += other.y; my.z += other.z; my.w += other.w;
                *reinterpret_cast<float4*>(smem + wid * num_datum + i) = my;
            }
        }
        __syncthreads();
    }
}

__device__ __forceinline__
void store_result_smem_rr(
    float Creg[2][4][4], float *smem_buf, float *C,
    const uint32_t M, const uint32_t N, const uint32_t cS,
    const uint32_t brid, const uint32_t bcid, const uint32_t tid, const uint32_t wid, const uint32_t lid,
    const uint32_t wcols, const uint32_t lrid, const uint32_t lcid
) {
    // 存在 slice_num 份矩阵 C 子区域的部分结果，需先使用共享内存对其进行归约   
    float *Csmem = smem_buf + 1024 * wid;
    // 写回矩阵 C 的子区域，使用 32x32 共享内存搬运 32x32 数据，共需 1 次
    float *C_block = C + (blockIdx.z * cS + brid * 32 * N + bcid * 32);

    float4 trans1, trans2;
    __syncthreads();
    // 首先，所有线程先将部分结果数据写入到共享内存，每个线程负责写回 Creg[2][4][4] 的数据
    #pragma unroll
    for (uint32_t row = 0; row < 4; ++row) {
        trans1.x = Creg[0][row][0]; trans1.y = Creg[0][row][1]; trans1.z = Creg[0][row][2]; trans1.w = Creg[0][row][3];
        trans2.x = Creg[1][row][0]; trans2.y = Creg[1][row][1]; trans2.z = Creg[1][row][2]; trans2.w = Creg[1][row][3];
        *reinterpret_cast<float4*>(Csmem + (0 * wcols * 4 + lcid * 4) + (lrid * 4 * 32 + row * 32)) = trans1;
        *reinterpret_cast<float4*>(Csmem + (1 * wcols * 4 + lcid * 4) + (lrid * 4 * 32 + row * 32)) = trans2;
    }
    __syncthreads();
    // 在 slice_num 个线程束之上进行归约，函数结束时存在显式 __syncthreads() 同步
    reduce_over_warp(smem_buf, 4, 1024, wid, lid);
    // 将数据从共享内存转移到全局内存
    // 使用 4x32 排列的线程搬运 32x32 共享内存，共需 8 次，每次每个线程写回 1 个数据
    #pragma unroll
    for (uint32_t gmem_row = 0; gmem_row < 32; gmem_row += 4) {
        if ((brid * 32 + gmem_row + tid / 32 < M) && (bcid * 32 + tid % 32 < N)) {
            *(C_block + (gmem_row + tid / 32) * N + (tid % 32)) = *(smem_buf + gmem_row * 32 + tid);
        }
    }
}

__device__ __forceinline__
void store_result_smem_rc(
    float Creg[2][4][4], float *smem_buf, float *C,
    const uint32_t M, const uint32_t N, const uint32_t cS,
    const uint32_t brid, const uint32_t bcid, const uint32_t tid, const uint32_t wid, const uint32_t lid,
    const uint32_t wcols, const uint32_t lrid, const uint32_t lcid
) {
    // 存在 slice_num 份矩阵 C 子区域的部分结果，需先使用共享内存对其进行归约   
    float *Csmem = smem_buf + 1024 * wid;
    // 写回矩阵 C 的子区域，使用 32x32 共享内存搬运 32x32 数据，共需 1 次
    float *C_block = C + (blockIdx.z * cS + bcid * 32 * M + brid * 32);

    float4 trans1, trans2;
    __syncthreads();
    // 首先，所有线程先将部分结果数据写入到共享内存，每个线程负责写回 Creg[2][4][4] 的数据
    #pragma unroll
    for (uint32_t column = 0; column < 4; ++column) {
        trans1.x = Creg[0][0][column]; trans1.y = Creg[0][1][column]; trans1.z = Creg[0][2][column]; trans1.w = Creg[0][3][column];
        trans2.x = Creg[1][0][column]; trans2.y = Creg[1][1][column]; trans2.z = Creg[1][2][column]; trans2.w = Creg[1][3][column];
        *reinterpret_cast<float4*>(Csmem + (0 * wcols * 4 * 32 + lcid * 4 * 32 + column * 32) + (lrid * 4)) = trans1;
        *reinterpret_cast<float4*>(Csmem + (1 * wcols * 4 * 32 + lcid * 4 * 32 + column * 32) + (lrid * 4)) = trans2;
    }
    __syncthreads();
    // 在 slice_num 个线程束之上进行归约，函数结束时存在显式 __syncthreads() 同步
    reduce_over_warp(smem_buf, 4, 1024, wid, lid);
    // 将数据从共享内存转移到全局内存
    // 使用 32x4 排列的线程搬运 32x32 共享内存，共需 8 次，每次每个线程写回 1 个数据
    #pragma unroll
    for (uint32_t gmem_column = 0; gmem_column < 32; gmem_column += 4) {

        if ((brid * 32 + tid % 32 < M) && (bcid * 32 + gmem_column + tid / 32 < N)) {
            *(C_block + (gmem_column + tid / 32) * M + (tid % 32)) = *(smem_buf + gmem_column * 32 + tid);
        }
    }
}

__device__ __forceinline__
void compute_tile_crr(
    float Creg[2][4][4], float *Asmem, float *Bsmem, const uint32_t ldA, const uint32_t ldB,
    const uint32_t wcols, const uint32_t lrid, const uint32_t lcid
) {
    float4 Areg, Breg[2];
    // 每个线程计算 C 的子域，采用向量外积方式，在 K_block 维度上循环迭代
    #pragma unroll
    for (uint32_t kid = 0; kid < 4; ++kid) {
        Areg = *reinterpret_cast<float4*>(Asmem + lrid * 4 + kid * ldA);
        Breg[0] = *reinterpret_cast<float4*>(Bsmem + 0 * wcols * 4 + lcid * 4 + kid * ldB);
        Breg[1] = *reinterpret_cast<float4*>(Bsmem + 1 * wcols * 4 + lcid * 4 + kid * ldB);
        #pragma unroll
        for (uint32_t cpj = 0; cpj < 2; ++cpj) {
            Creg[cpj][0][0] += Areg.x * Breg[cpj].x;
            Creg[cpj][0][1] += Areg.x * Breg[cpj].y;
            Creg[cpj][0][2] += Areg.x * Breg[cpj].z;
            Creg[cpj][0][3] += Areg.x * Breg[cpj].w;
            Creg[cpj][1][0] += Areg.y * Breg[cpj].x;
            Creg[cpj][1][1] += Areg.y * Breg[cpj].y;
            Creg[cpj][1][2] += Areg.y * Breg[cpj].z;
            Creg[cpj][1][3] += Areg.y * Breg[cpj].w;
            Creg[cpj][2][0] += Areg.z * Breg[cpj].x;
            Creg[cpj][2][1] += Areg.z * Breg[cpj].y;
            Creg[cpj][2][2] += Areg.z * Breg[cpj].z;
            Creg[cpj][2][3] += Areg.z * Breg[cpj].w;
            Creg[cpj][3][0] += Areg.w * Breg[cpj].x;
            Creg[cpj][3][1] += Areg.w * Breg[cpj].y;
            Creg[cpj][3][2] += Areg.w * Breg[cpj].z;
            Creg[cpj][3][3] += Areg.w * Breg[cpj].w;
        }
    }
}

__device__ __forceinline__
void compute_block_rrr(
    float Creg[2][4][4], float *smem_buf, const float *A, const float *B, const float alpha, const ShapeLayoutSliceK &SL
) {
    float *Asmem = smem_buf + 1024 * SL.wid;
    float *Bsmem = smem_buf + 1024 * SL.wid + (128 + 32) * 2;

    // [NEXT] A_lid + eid * SL.K + kth * 16       + slice_idx * 4
    // [NEXT] B_lid + eid * SL.N + kth * 16 * SL.N + slice_idx * 4 * SL.N
    const float *A_lid = A + (blockIdx.z * SL.aS + SL.brid * 32 * SL.K) + (SL.lid / 4 * 4 * SL.K + SL.lid % 4);
    const float *B_lid = B + (blockIdx.z * SL.bS + SL.bcid * 32) + SL.lid;
    float Atrans[4] = {}, Btrans[4] = {};

    // valid[eid] 标识 eid 数据是否为有效数据，有效元素指未越界的数据
    uint32_t A_valid = 0U, B_valid = 0U;
    #pragma unroll
    for (uint32_t eid = 0; eid < 4; ++eid) {
        if (SL.brid * 32 + SL.lid / 4 * 4 + eid < SL.M) A_valid |= (1u << eid);
        if (SL.bcid * 32 + SL.lid < SL.N)               B_valid |= (1u << eid);
    }

    // 一次完整的 slice_num = 4 迭代在 K 的维度上读取 slice_num * slice_len = 4 * 4 = 16 的数据，首先处理刚开始的可能情况
    uint32_t kstart = SL.K - ((SL.K + 15) / 16 - 1) * 16;  // [1, 2, 3, ..., 16]
    // 预取可能不足 16 个的元素
    #pragma unroll
    for (uint32_t eid = 0; eid < 4; ++eid) {
        if ((A_valid & (1u << eid)) && (SL.wid * 4 + SL.lid % 4 < kstart)) {
            Atrans[eid] = *(A_lid + SL.wid * 4 + eid * SL.K);
        }
        if ((B_valid & (1u << eid)) && (SL.wid * 4 + eid < kstart)) {
            Btrans[eid] = *(B_lid + SL.wid * 4 * SL.N + eid * SL.N);
        }
    }

    // 将预取数据写入到共享内存
    // 此处采用 32 + 4 是因为使用 4 做偏移时，保证可使用 float4 向量化读写共享内存，且使用 float4 写入时不存在 bank 冲突
    *reinterpret_cast<float4*>(Asmem + SL.lid % 4 * 36 + SL.lid / 4 * 4) = *reinterpret_cast<float4*>(Atrans);
    Bsmem[SL.lid + 0 * 32] = Btrans[0];
    Bsmem[SL.lid + 1 * 32] = Btrans[1];
    Bsmem[SL.lid + 2 * 32] = Btrans[2];
    Bsmem[SL.lid + 3 * 32] = Btrans[3];
    __syncthreads();
    A_lid += kstart;
    B_lid += kstart * SL.N;

    // 在 K 的维度轴上进行循环迭代，计算矩阵 C 的子区域
    for (uint32_t kth = 1; kth < (SL.K + 15) / 16; ++kth) {
        // 预取 kth 的数据
        #pragma unroll
        for (uint32_t eid = 0; eid < 4; ++eid) {
            if (A_valid & (1u << eid)) {
                Atrans[eid] = *(A_lid + SL.wid * 4 + eid * SL.K);
            }
            if (B_valid & (1u << eid)) {
                Btrans[eid] = *(B_lid + SL.wid * 4 * SL.N + eid * SL.N);
            }
        }
        // 计算 C 的子区域
        compute_tile_crr(Creg, Asmem, Bsmem, 36, 32, SL.wcols, SL.lrid, SL.lcid);
        // 将预取数据写入到共享内存
        Asmem += (2 * (kth & 1) - 1) * (128 + 32);
        Bsmem += (2 * (kth & 1) - 1) * 128;
        *reinterpret_cast<float4*>(Asmem + SL.lid % 4 * 36 + SL.lid / 4 * 4) = *reinterpret_cast<float4*>(Atrans);
        Bsmem[SL.lid + 0 * 32] = Btrans[0];
        Bsmem[SL.lid + 1 * 32] = Btrans[1];
        Bsmem[SL.lid + 2 * 32] = Btrans[2];
        Bsmem[SL.lid + 3 * 32] = Btrans[3];
        __syncthreads();
        A_lid += 16;
        B_lid += 16 * SL.N;
    }
    // 计算 C 的子区域
    compute_tile_crr(Creg, Asmem, Bsmem, 36, 32, SL.wcols, SL.lrid, SL.lcid);

    // 应用 alpha 缩放
    #pragma unroll
    for (uint32_t cpj = 0; cpj < 2; ++cpj) {
        #pragma unroll
        for (uint32_t row = 0; row < 4; ++row) {
            Creg[cpj][row][0] *= alpha;
            Creg[cpj][row][1] *= alpha;
            Creg[cpj][row][2] *= alpha;
            Creg[cpj][row][3] *= alpha;
        }
    }
}

__device__ __forceinline__
void compute_block_rcr(
    float Creg[2][4][4], float *smem_buf, const float *A, const float *B, const float alpha, const ShapeLayoutSliceK &SL
) {
    float *Asmem = smem_buf + 1024 * SL.wid;
    float *Bsmem = smem_buf + 1024 * SL.wid + (128 + 32) * 2;

    // [NEXT] A_lid + eid * SL.K + kth * 16 + slice_idx * 4
    // [NEXT] B_lid + eid * SL.K + kth * 16 + slice_idx * 4
    const float *A_lid = A + (blockIdx.z * SL.aS + SL.brid * 32 * SL.K) + (SL.lid / 4 * 4 * SL.K + SL.lid % 4);
    const float *B_lid = B + (blockIdx.z * SL.bS + SL.bcid * 32 * SL.K) + (SL.lid / 4 * 4 * SL.K + SL.lid % 4);
    float Atrans[4] = {}, Btrans[4] = {};

    // valid[eid] 标识 eid 数据是否为有效数据，有效元素指未越界的数据
    uint32_t A_valid = 0U, B_valid = 0U;
    #pragma unroll
    for (uint32_t eid = 0; eid < 4; ++eid) {
        if (SL.brid * 32 + SL.lid / 4 * 4 + eid < SL.M) A_valid |= (1u << eid);
        if (SL.bcid * 32 + SL.lid / 4 * 4 + eid < SL.N) B_valid |= (1u << eid);
    }

    // 一次完整的 slice_num = 4 迭代在 K 的维度上读取 slice_num * slice_len = 4 * 4 = 16 的数据，首先处理刚开始的可能情况
    uint32_t kstart = SL.K - ((SL.K + 15) / 16 - 1) * 16;  // [1, 2, 3, ..., 16]
    // 预取可能不足 16 个的元素
    #pragma unroll
    for (uint32_t eid = 0; eid < 4; ++eid) {
        if ((A_valid & (1u << eid)) && (SL.wid * 4 + SL.lid % 4 < kstart)) {
            Atrans[eid] = *(A_lid + SL.wid * 4 + eid * SL.K);
        }
        if ((B_valid & (1u << eid)) && (SL.wid * 4 + SL.lid % 4 < kstart)) {
            Btrans[eid] = *(B_lid + SL.wid * 4 + eid * SL.K);
        }
    }

    // 将预取数据写入到共享内存
    // 此处采用 32 + 4 是因为使用 4 做偏移时，保证可使用 float4 向量化读写共享内存，且使用 float4 写入时不存在 bank 冲突
    *reinterpret_cast<float4*>(Asmem + SL.lid % 4 * 36 + SL.lid / 4 * 4) = *reinterpret_cast<float4*>(Atrans);
    *reinterpret_cast<float4*>(Bsmem + SL.lid % 4 * 36 + SL.lid / 4 * 4) = *reinterpret_cast<float4*>(Btrans);
    __syncthreads();
    A_lid += kstart;
    B_lid += kstart;

    // 在 K 的维度轴上进行循环迭代，计算矩阵 C 的子区域
    for (uint32_t kth = 1; kth < (SL.K + 15) / 16; ++kth) {
        // 预取 kth 的数据
        #pragma unroll
        for (uint32_t eid = 0; eid < 4; ++eid) {
            if (A_valid & (1u << eid)) {
                Atrans[eid] = *(A_lid + SL.wid * 4 + eid * SL.K);
            }
            if (B_valid & (1u << eid)) {
                Btrans[eid] = *(B_lid + SL.wid * 4 + eid * SL.K);
            }
        }
        // 计算 C 的子区域
        compute_tile_crr(Creg, Asmem, Bsmem, 36, 36, SL.wcols, SL.lrid, SL.lcid);
        // 将预取数据写入到共享内存
        Asmem += (2 * (kth & 1) - 1) * (128 + 32);
        Bsmem += (2 * (kth & 1) - 1) * (128 + 32);
        *reinterpret_cast<float4*>(Asmem + SL.lid % 4 * 36 + SL.lid / 4 * 4) = *reinterpret_cast<float4*>(Atrans);
        *reinterpret_cast<float4*>(Bsmem + SL.lid % 4 * 36 + SL.lid / 4 * 4) = *reinterpret_cast<float4*>(Btrans);
        __syncthreads();
        A_lid += 16;
        B_lid += 16;
    }
    // 计算 C 的子区域
    compute_tile_crr(Creg, Asmem, Bsmem, 36, 36, SL.wcols, SL.lrid, SL.lcid);

    // 应用 alpha 缩放
    #pragma unroll
    for (uint32_t cpj = 0; cpj < 2; ++cpj) {
        #pragma unroll
        for (uint32_t row = 0; row < 4; ++row) {
            Creg[cpj][row][0] *= alpha;
            Creg[cpj][row][1] *= alpha;
            Creg[cpj][row][2] *= alpha;
            Creg[cpj][row][3] *= alpha;
        }
    }
}

__device__ __forceinline__
void compute_block_crr(
    float Creg[2][4][4], float *smem_buf, const float *A, const float *B, const float alpha, const ShapeLayoutSliceK &SL
) {
    float *Asmem = smem_buf + 1024 * SL.wid;
    float *Bsmem = smem_buf + 1024 * SL.wid + 128 * 2;

    // [NEXT] A_lid + eid * SL.M + kth * 16 * SL.M + slice_idx * 4 * SL.M
    // [NEXT] B_lid + eid * SL.N + kth * 16 * SL.N + slice_idx * 4 * SL.N
    const float *A_lid = A + (blockIdx.z * SL.aS + SL.brid * 32) + SL.lid;
    const float *B_lid = B + (blockIdx.z * SL.bS + SL.bcid * 32) + SL.lid;
    float Atrans[4] = {}, Btrans[4] = {};

    // valid[eid] 标识 eid 数据是否为有效数据，有效元素指未越界的数据
    uint32_t A_valid = 0U, B_valid = 0U;
    #pragma unroll
    for (uint32_t eid = 0; eid < 4; ++eid) {
        if (SL.brid * 32 + SL.lid < SL.M) A_valid |= (1u << eid);
        if (SL.bcid * 32 + SL.lid < SL.N) B_valid |= (1u << eid);
    }

    // 一次完整的 slice_num = 4 迭代在 K 的维度上读取 slice_num * slice_len = 4 * 4 = 16 的数据，首先处理刚开始的可能情况
    uint32_t kstart = SL.K - ((SL.K + 15) / 16 - 1) * 16;  // [1, 2, 3, ..., 16]
    // 预取可能不足 16 个的元素
    #pragma unroll
    for (uint32_t eid = 0; eid < 4; ++eid) {
        if ((A_valid & (1u << eid)) && (SL.wid * 4 + eid < kstart)) {
            Atrans[eid] = *(A_lid + SL.wid * 4 * SL.M + eid * SL.M);
        }
        if ((B_valid & (1u << eid)) && (SL.wid * 4 + eid < kstart)) {
            Btrans[eid] = *(B_lid + SL.wid * 4 * SL.N + eid * SL.N);
        }
    }

    // 将预取数据写入到共享内存
    Asmem[SL.lid + 0 * 32] = Atrans[0];
    Asmem[SL.lid + 1 * 32] = Atrans[1];
    Asmem[SL.lid + 2 * 32] = Atrans[2];
    Asmem[SL.lid + 3 * 32] = Atrans[3];
    Bsmem[SL.lid + 0 * 32] = Btrans[0];
    Bsmem[SL.lid + 1 * 32] = Btrans[1];
    Bsmem[SL.lid + 2 * 32] = Btrans[2];
    Bsmem[SL.lid + 3 * 32] = Btrans[3];
    __syncthreads();
    A_lid += kstart * SL.M;
    B_lid += kstart * SL.N;

    // 在 K 的维度轴上进行循环迭代，计算矩阵 C 的子区域
    for (uint32_t kth = 1; kth < (SL.K + 15) / 16; ++kth) {
        // 预取 kth 的数据
        #pragma unroll
        for (uint32_t eid = 0; eid < 4; ++eid) {
            if (A_valid & (1u << eid)) {
                Atrans[eid] = *(A_lid + SL.wid * 4 * SL.M + eid * SL.M);
            }
            if (B_valid & (1u << eid)) {
                Btrans[eid] = *(B_lid + SL.wid * 4 * SL.N + eid * SL.N);
            }
        }
        // 计算 C 的子区域
        compute_tile_crr(Creg, Asmem, Bsmem, 32, 32, SL.wcols, SL.lrid, SL.lcid);
        // 将预取数据写入到共享内存
        Asmem += (2 * (kth & 1) - 1) * 128;
        Bsmem += (2 * (kth & 1) - 1) * 128;
        Asmem[SL.lid + 0 * 32] = Atrans[0];
        Asmem[SL.lid + 1 * 32] = Atrans[1];
        Asmem[SL.lid + 2 * 32] = Atrans[2];
        Asmem[SL.lid + 3 * 32] = Atrans[3];
        Bsmem[SL.lid + 0 * 32] = Btrans[0];
        Bsmem[SL.lid + 1 * 32] = Btrans[1];
        Bsmem[SL.lid + 2 * 32] = Btrans[2];
        Bsmem[SL.lid + 3 * 32] = Btrans[3];
        __syncthreads();
        A_lid += 16 * SL.M;
        B_lid += 16 * SL.N;
    }
    // 计算 C 的子区域
    compute_tile_crr(Creg, Asmem, Bsmem, 32, 32, SL.wcols, SL.lrid, SL.lcid);

    // 应用 alpha 缩放
    #pragma unroll
    for (uint32_t cpj = 0; cpj < 2; ++cpj) {
        #pragma unroll
        for (uint32_t row = 0; row < 4; ++row) {
            Creg[cpj][row][0] *= alpha;
            Creg[cpj][row][1] *= alpha;
            Creg[cpj][row][2] *= alpha;
            Creg[cpj][row][3] *= alpha;
        }
    }
}

__device__ __forceinline__
void compute_block_ccr(
    float Creg[2][4][4], float *smem_buf, const float *A, const float *B, const float alpha, const ShapeLayoutSliceK &SL
) {
    float *Asmem = smem_buf + 1024 * SL.wid;
    float *Bsmem = smem_buf + 1024 * SL.wid + 128 * 2;

    // [NEXT] A_lid + eid * SL.M + kth * 16 * SL.M + slice_idx * 4 * SL.M
    // [NEXT] B_lid + eid * SL.K + kth * 16 + slice_idx * 4
    const float *A_lid = A + (blockIdx.z * SL.aS + SL.brid * 32) + SL.lid;
    const float *B_lid = B + (blockIdx.z * SL.bS + SL.bcid * 32 * SL.K) + (SL.lid / 4 * 4 * SL.K + SL.lid % 4);
    float Atrans[4] = {}, Btrans[4] = {};

    // valid[eid] 标识 eid 数据是否为有效数据，有效元素指未越界的数据
    uint32_t A_valid = 0U, B_valid = 0U;
    #pragma unroll
    for (uint32_t eid = 0; eid < 4; ++eid) {
        if (SL.brid * 32 + SL.lid < SL.M)               A_valid |= (1u << eid);
        if (SL.bcid * 32 + SL.lid / 4 * 4 + eid < SL.N) B_valid |= (1u << eid);
    }

    // 一次完整的 slice_num = 4 迭代在 K 的维度上读取 slice_num * slice_len = 4 * 4 = 16 的数据，首先处理刚开始的可能情况
    uint32_t kstart = SL.K - ((SL.K + 15) / 16 - 1) * 16;  // [1, 2, 3, ..., 16]
    // 预取可能不足 16 个的元素
    #pragma unroll
    for (uint32_t eid = 0; eid < 4; ++eid) {
        if ((A_valid & (1u << eid)) && (SL.wid * 4 + eid < kstart)) {
            Atrans[eid] = *(A_lid + SL.wid * 4 * SL.M + eid * SL.M);
        }
        if ((B_valid & (1u << eid)) && (SL.wid * 4 + SL.lid % 4 < kstart)) {
            Btrans[eid] = *(B_lid + SL.wid * 4 + eid * SL.K);
        }
    }

    // 将预取数据写入到共享内存
    Asmem[SL.lid + 0 * 32] = Atrans[0];
    Asmem[SL.lid + 1 * 32] = Atrans[1];
    Asmem[SL.lid + 2 * 32] = Atrans[2];
    Asmem[SL.lid + 3 * 32] = Atrans[3];
    // 此处采用 32 + 4 是因为使用 4 做偏移时，保证可使用 float4 向量化读写共享内存，且使用 float4 写入时不存在 bank 冲突
    *reinterpret_cast<float4*>(Bsmem + SL.lid % 4 * 36 + SL.lid / 4 * 4) = *reinterpret_cast<float4*>(Btrans);
    __syncthreads();
    A_lid += kstart * SL.M;
    B_lid += kstart;

    // 在 K 的维度轴上进行循环迭代，计算矩阵 C 的子区域
    for (uint32_t kth = 1; kth < (SL.K + 15) / 16; ++kth) {
        // 预取 kth 的数据
        #pragma unroll
        for (uint32_t eid = 0; eid < 4; ++eid) {
            if (A_valid & (1u << eid)) {
                Atrans[eid] = *(A_lid + SL.wid * 4 * SL.M + eid * SL.M);
            }
            if (B_valid & (1u << eid)) {
                Btrans[eid] = *(B_lid + SL.wid * 4 + eid * SL.K);
            }
        }
        // 计算 C 的子区域
        compute_tile_crr(Creg, Asmem, Bsmem, 32, 36, SL.wcols, SL.lrid, SL.lcid);
        // 将预取数据写入到共享内存
        Asmem += (2 * (kth & 1) - 1) * 128;
        Bsmem += (2 * (kth & 1) - 1) * (128 + 32);
        Asmem[SL.lid + 0 * 32] = Atrans[0];
        Asmem[SL.lid + 1 * 32] = Atrans[1];
        Asmem[SL.lid + 2 * 32] = Atrans[2];
        Asmem[SL.lid + 3 * 32] = Atrans[3];
        // 此处采用 32 + 4 是因为使用 4 做偏移时，保证可使用 float4 向量化读写共享内存，且使用 float4 写入时不存在 bank 冲突
        *reinterpret_cast<float4*>(Bsmem + SL.lid % 4 * 36 + SL.lid / 4 * 4) = *reinterpret_cast<float4*>(Btrans);
        __syncthreads();
        A_lid += 16 * SL.M;
        B_lid += 16;
    }
    // 计算 C 的子区域
    compute_tile_crr(Creg, Asmem, Bsmem, 32, 36, SL.wcols, SL.lrid, SL.lcid);

    // 应用 alpha 缩放
    #pragma unroll
    for (uint32_t cpj = 0; cpj < 2; ++cpj) {
        #pragma unroll
        for (uint32_t row = 0; row < 4; ++row) {
            Creg[cpj][row][0] *= alpha;
            Creg[cpj][row][1] *= alpha;
            Creg[cpj][row][2] *= alpha;
            Creg[cpj][row][3] *= alpha;
        }
    }
}

__global__ void sgemm_rrr_kernel(
    const float *A, const float *B, float *C, const float alpha,
    const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS
) {
    float *smem_buf = buffer::SharedMemory<float, 1024 * 4>().pointer();
    ShapeLayoutSliceK SL(M, N, K, aS, bS, cS);
    float Creg[2][4][4] = {};
    compute_block_rrr(Creg, smem_buf, A, B, alpha, SL);
    store_result_smem_rr(Creg, smem_buf, C, SL.M, SL.N, SL.cS, SL.brid, SL.bcid, SL.tid, SL.wid, SL.lid, SL.wcols, SL.lrid, SL.lcid);
}

__global__ void sgemm_rrc_kernel(
    const float *A, const float *B, float *C, const float alpha,
    const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS
) {
    float *smem_buf = buffer::SharedMemory<float, 1024 * 4>().pointer();
    ShapeLayoutSliceK SL(M, N, K, aS, bS, cS);
    float Creg[2][4][4] = {};
    compute_block_rrr(Creg, smem_buf, A, B, alpha, SL);
    store_result_smem_rc(Creg, smem_buf, C, SL.M, SL.N, SL.cS, SL.brid, SL.bcid, SL.tid, SL.wid, SL.lid, SL.wcols, SL.lrid, SL.lcid);
}

__global__ void sgemm_rcr_kernel(
    const float *A, const float *B, float *C, const float alpha,
    const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS
) {
    float *smem_buf = buffer::SharedMemory<float, 1024 * 4>().pointer();
    ShapeLayoutSliceK SL(M, N, K, aS, bS, cS);
    float Creg[2][4][4] = {};
    compute_block_rcr(Creg, smem_buf, A, B, alpha, SL);
    store_result_smem_rr(Creg, smem_buf, C, SL.M, SL.N, SL.cS, SL.brid, SL.bcid, SL.tid, SL.wid, SL.lid, SL.wcols, SL.lrid, SL.lcid);
}

__global__ void sgemm_rcc_kernel(
    const float *A, const float *B, float *C, const float alpha,
    const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS
) {
    float *smem_buf = buffer::SharedMemory<float, 1024 * 4>().pointer();
    ShapeLayoutSliceK SL(M, N, K, aS, bS, cS);
    float Creg[2][4][4] = {};
    compute_block_rcr(Creg, smem_buf, A, B, alpha, SL);
    store_result_smem_rc(Creg, smem_buf, C, SL.M, SL.N, SL.cS, SL.brid, SL.bcid, SL.tid, SL.wid, SL.lid, SL.wcols, SL.lrid, SL.lcid);
}

__global__ void sgemm_crr_kernel(
    const float *A, const float *B, float *C, const float alpha,
    const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS
) {
    float *smem_buf = buffer::SharedMemory<float, 1024 * 4>().pointer();
    ShapeLayoutSliceK SL(M, N, K, aS, bS, cS);
    float Creg[2][4][4] = {};
    compute_block_crr(Creg, smem_buf, A, B, alpha, SL);
    store_result_smem_rr(Creg, smem_buf, C, SL.M, SL.N, SL.cS, SL.brid, SL.bcid, SL.tid, SL.wid, SL.lid, SL.wcols, SL.lrid, SL.lcid);
}

__global__ void sgemm_crc_kernel(
    const float *A, const float *B, float *C, const float alpha,
    const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS
) {
    float *smem_buf = buffer::SharedMemory<float, 1024 * 4>().pointer();
    ShapeLayoutSliceK SL(M, N, K, aS, bS, cS);
    float Creg[2][4][4] = {};
    compute_block_crr(Creg, smem_buf, A, B, alpha, SL);
    store_result_smem_rc(Creg, smem_buf, C, SL.M, SL.N, SL.cS, SL.brid, SL.bcid, SL.tid, SL.wid, SL.lid, SL.wcols, SL.lrid, SL.lcid);
}

__global__ void sgemm_ccr_kernel(
    const float *A, const float *B, float *C, const float alpha,
    const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS
) {
    float *smem_buf = buffer::SharedMemory<float, 1024 * 4>().pointer();
    ShapeLayoutSliceK SL(M, N, K, aS, bS, cS);
    float Creg[2][4][4] = {};
    compute_block_ccr(Creg, smem_buf, A, B, alpha, SL);
    store_result_smem_rr(Creg, smem_buf, C, SL.M, SL.N, SL.cS, SL.brid, SL.bcid, SL.tid, SL.wid, SL.lid, SL.wcols, SL.lrid, SL.lcid);
}

__global__ void sgemm_ccc_kernel(
    const float *A, const float *B, float *C, const float alpha,
    const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS
) {
    float *smem_buf = buffer::SharedMemory<float, 1024 * 4>().pointer();
    ShapeLayoutSliceK SL(M, N, K, aS, bS, cS);
    float Creg[2][4][4] = {};
    compute_block_ccr(Creg, smem_buf, A, B, alpha, SL);
    store_result_smem_rc(Creg, smem_buf, C, SL.M, SL.N, SL.cS, SL.brid, SL.bcid, SL.tid, SL.wid, SL.lid, SL.wcols, SL.lrid, SL.lcid);
}

__host__ void sgemm_cuda(
    const float *A, const float *B, float *C, const float alpha,
    const uint32_t M, const uint32_t N, const uint32_t K, const uint32_t aS, const uint32_t bS, const uint32_t cS,
    const GEMM_Order order, const uint32_t batchCount
) {
    const dim3 block_size(128, 1, 1);
    const dim3 grid_size((N + 31) / 32, (M + 31) / 32, batchCount);
    switch (order) {
    case GEMM_Order::RRR:
        sgemm_rrr_kernel<<<grid_size, block_size>>>(A, B, C, alpha, M, N, K, aS, bS, cS); break;
    case GEMM_Order::RRC:
        sgemm_rrc_kernel<<<grid_size, block_size>>>(A, B, C, alpha, M, N, K, aS, bS, cS); break;
    case GEMM_Order::RCR:
        sgemm_rcr_kernel<<<grid_size, block_size>>>(A, B, C, alpha, M, N, K, aS, bS, cS); break;
    case GEMM_Order::RCC:
        sgemm_rcc_kernel<<<grid_size, block_size>>>(A, B, C, alpha, M, N, K, aS, bS, cS); break;
    case GEMM_Order::CRR:
        sgemm_crr_kernel<<<grid_size, block_size>>>(A, B, C, alpha, M, N, K, aS, bS, cS); break;
    case GEMM_Order::CRC:
        sgemm_crc_kernel<<<grid_size, block_size>>>(A, B, C, alpha, M, N, K, aS, bS, cS); break;
    case GEMM_Order::CCR:
        sgemm_ccr_kernel<<<grid_size, block_size>>>(A, B, C, alpha, M, N, K, aS, bS, cS); break;
    case GEMM_Order::CCC:
        sgemm_ccc_kernel<<<grid_size, block_size>>>(A, B, C, alpha, M, N, K, aS, bS, cS); break;
    default: break;
    }
}

} // namespace sgemm_32x32_4x8_SliceK
