#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_runtime.h>
#include "../utils/buffer.cu"
#include "../utils/ptx.cu"

/**
 * Matrix A, B, C : row-major
 * Threadblock Tile : [M, N, K] = [128, 128, 8]
 * Warp Tile : [M, N, K] = [32, 64, 8]
 * Thread Tile : [M, N, K] = [8, 8, 8]
 * A_tile and B_tile : [128, 8] and [8, 128]
 * A_frag and B_frag : [8, 1] and [1, 8]
 */
__global__ __launch_bounds__(256, 2)
void sgemm_rrr_128x128x8_kernel(
    const float *A, const float *B, float *C,
    const uint32_t M, const uint32_t N, const uint32_t K
) {
    // A and B Threadblock Tile on shared memory (double buffer)
    // A_tile : 132 * 8 * float * double buffer = 4.125 KiB * 2
    // B_tile : 128 * 8 * float * double buffer = 4 KiB * 2
    // 为更快地切换 A_tile 和 B_tile 的缓冲区，
    // A_tile 需要一块连续的 8 KiB * 2 = 2^13 B * 2 的缓冲区，故可以使用 (uint32_t&) A_smem ^= 0x2000; 进行切换
    // B_tile 需要一块连续的 4 KiB * 2 = 2^12 B * 2 的缓冲区，故可以使用 (uint32_t&) B_smem ^= 0x1000; 进行切换
    // 如此，共享内存双缓冲的切换，只需要使用一条异或指令即可
    float *smem_buf = buffer::SharedMemory<float, 128 * 8 * 6>().pointer();
    float *A_smem = reinterpret_cast<float*>(smem_buf);
    float *B_smem = reinterpret_cast<float*>(smem_buf + 128 * 8 * 4);

    // A, B Thread Tile on register, C Thread Tile on register (double buffer)
    float A_frag[2][8], B_frag[2][8], C_frag[8][8] = { 0 };

    // 一个Warp中的线程标识，排列成 4x8 形状
    const uint32_t warp_id = threadIdx.x / 32;
    const uint32_t lane_id = threadIdx.x % 32;
    const uint32_t lane_rid = (lane_id / 16) * 2 + (lane_id % 2);
    const uint32_t lane_cid = (lane_id / 2) % 8;

    // A_tile and B_tile ldg pointer, Threadblock arranged as row-major
    // [NEXT] = A_ldg_ptr + K_tile;      [eid] = A_ldg_ptr + eid * K;
    // [NEXT] = B_ldg_ptr + K_tile * N;  [eid] = B_ldg_ptr + eid * 32;
    const float *A_ldg_ptr = reinterpret_cast<const float*>(A + blockIdx.y * 128 * K + threadIdx.x / 8 * 4 * K + threadIdx.x % 8);
    const float *B_ldg_ptr = reinterpret_cast<const float*>(B + blockIdx.x * 128 + threadIdx.x / 32 * N + threadIdx.x % 32);

    // A_tile and B_tile sts address
    // [eid] = A_sts_addr + eid * sizeof(float)
    // [eid] = B_sts_addr + eid * 32 * sizeof(float)
    uint32_t A_sts_addr = ptx::smem_addr(A_smem + threadIdx.x % 8 * 132 + threadIdx.x / 8 * 4);
    uint32_t B_sts_addr = ptx::smem_addr(B_smem + threadIdx.x / 32 * 128 + threadIdx.x % 32);

    // A_tile and B_tile lds address, four sub-partitions: [0][0], [0][1], [1][0], [1][1]
    // [eid] = A_lds_addr + eid * 132 * sizeof(float);  [prid][pcid] = A_lds_addr + prid * 4 * 4 * sizeof(float)
    // [eid] = B_lds_addr + eid * 128 * sizeof(float);  [prid][pcid] = B_lds_addr + pcid * 8 * 4 * sizeof(float)
    uint32_t A_lds_addr = ptx::smem_addr(A_smem + warp_id / 2 * 32 + lane_rid * 4);
    uint32_t B_lds_addr = ptx::smem_addr(B_smem + warp_id % 2 * 64 + lane_cid * 4);

    // ldg_valid[eid] 标识 eid 数据是否为有效数据，有效元素指未越界的数据，避免 ldg 指令越界
    uint32_t A_ldg_valid = 0, B_ldg_valid = 0;
    #pragma unroll
    for (uint32_t eid = 0; eid < 4; ++eid) {
        A_ldg_valid |= (uint32_t)(blockIdx.y * 128 + threadIdx.x / 8 * 4 + eid < M)   << eid;
        B_ldg_valid |= (uint32_t)(blockIdx.x * 128 + threadIdx.x % 32 + eid * 32 < N) << eid;
    }

    // A, B ldg buffer for transfering data from gmem to smem
    float A_ldg_buf[4], B_ldg_buf[4];

    // the first A_tile and B_tile load before K-Loop, handling boundary (maybe not 8 data)
    {
        uint32_t first_k_tile = K - ((K + 7) / 8 - 1) * 8;
        #pragma unroll
        for (uint32_t eid = 0; eid < 4; ++eid) {
            ptx::ld_gmem_zero(A_ldg_buf[eid], A_ldg_ptr + eid * K, (A_ldg_valid & (1u << eid)) && threadIdx.x % 8 < first_k_tile);
        }
        ptx::st_smem(A_ldg_buf[0], A_ldg_buf[1], A_ldg_buf[2], A_ldg_buf[3], A_sts_addr);
        #pragma unroll
        for (uint32_t eid = 0; eid < 4; ++eid) {
            ptx::ld_gmem_zero(B_ldg_buf[eid], B_ldg_ptr + eid * 32, (B_ldg_valid & (1u << eid)) && threadIdx.x / 32 < first_k_tile);
        }
        #pragma unroll
        for (uint32_t eid = 0; eid < 4; ++eid) {
            ptx::st_smem(B_ldg_buf[eid], B_sts_addr + eid * 32 * sizeof(float));
        }
        __syncthreads();
        // switch double buffer
        A_sts_addr ^= 0x2000;
        B_sts_addr ^= 0x1000;
        // ldg pointer for next tile
        A_ldg_ptr += first_k_tile;
        B_ldg_ptr += first_k_tile * N;
    }

    // load the first fragment
    ptx::ld_smem(A_frag[0][0], A_frag[0][1], A_frag[0][2], A_frag[0][3], A_lds_addr);
    ptx::ld_smem(A_frag[0][4], A_frag[0][5], A_frag[0][6], A_frag[0][7], A_lds_addr + 16 * sizeof(float));
    ptx::ld_smem(B_frag[0][0], B_frag[0][1], B_frag[0][2], B_frag[0][3], B_lds_addr);
    ptx::ld_smem(B_frag[0][4], B_frag[0][5], B_frag[0][6], B_frag[0][7], B_lds_addr + 32 * sizeof(float));

    // K-Loop, and K_tile is 8
    for (uint32_t num_k_tiles = (K + 7) / 8 - 1; num_k_tiles > 0; --num_k_tiles) {
        #pragma unroll
        for (int k_frag = 0; k_frag < 8; ++k_frag) {
            // K_tile 次计算即将执行完毕，将下一个 A_tile 和 B_tile 写入共享内存
            if (k_frag == 7) {
                ptx::st_smem(A_ldg_buf[0], A_ldg_buf[1], A_ldg_buf[2], A_ldg_buf[3], A_sts_addr);
                #pragma unroll
                for (uint32_t eid = 0; eid < 4; ++eid) {
                    ptx::st_smem(B_ldg_buf[eid], B_sts_addr + eid * 32 * sizeof(float));
                }
                __syncthreads();
                // switch double buffer
                A_sts_addr ^= 0x2000;
                B_sts_addr ^= 0x1000;
                A_lds_addr ^= 0x2000;
                B_lds_addr ^= 0x1000;
                // ldg pointer for next tile
                A_ldg_ptr += 8;
                B_ldg_ptr += 8 * N;
            }
            // 读取下一次计算所需的 A_frag 和 B_frag 并写入寄存器
            ptx::ld_smem(
                A_frag[(k_frag + 1) % 2][0], A_frag[(k_frag + 1) % 2][1],
                A_frag[(k_frag + 1) % 2][2], A_frag[(k_frag + 1) % 2][3],
                A_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float)
            );
            ptx::ld_smem(
                A_frag[(k_frag + 1) % 2][4], A_frag[(k_frag + 1) % 2][5],
                A_frag[(k_frag + 1) % 2][6], A_frag[(k_frag + 1) % 2][7],
                A_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float)
            );
            ptx::ld_smem(
                B_frag[(k_frag + 1) % 2][0], B_frag[(k_frag + 1) % 2][1],
                B_frag[(k_frag + 1) % 2][2], B_frag[(k_frag + 1) % 2][3],
                B_lds_addr + (k_frag + 1) % 8 * 128 * sizeof(float)
            );
            ptx::ld_smem(
                B_frag[(k_frag + 1) % 2][4], B_frag[(k_frag + 1) % 2][5],
                B_frag[(k_frag + 1) % 2][6], B_frag[(k_frag + 1) % 2][7],
                B_lds_addr + ((k_frag + 1) % 8 * 128 + 32) * sizeof(float)
            );
            // K_tile 的第一次计算之前，读取下一个 A_tile 和 B_tile 数据
            if (k_frag == 0) {
                #pragma unroll
                for (uint32_t eid = 0; eid < 4; ++eid) {
                    ptx::ld_gmem_zero(A_ldg_buf[eid], A_ldg_ptr + eid * K, A_ldg_valid & (1u << eid));
                }
                #pragma unroll
                for (uint32_t eid = 0; eid < 4; ++eid) {
                    ptx::ld_gmem_zero(B_ldg_buf[eid], B_ldg_ptr + eid * 32, B_ldg_valid & (1u << eid));
                }
            }
            // 执行FFMA计算
            #pragma unroll
            for (uint32_t i = 0; i < 8; ++i) {
                #pragma unroll
                for (uint32_t j = 0; j < 8; ++j) {
                    C_frag[i][j] += A_frag[k_frag % 2][i] * B_frag[k_frag % 2][j];
                }
            }
        }
    }
    // 最后一个 tile 的迭代
    #pragma unroll
    for (int k_frag = 0; k_frag < 8; ++k_frag) {
        // 读取下一次计算所需的 A_frag 和 B_frag 并写入寄存器
        if (k_frag < 7) {
            ptx::ld_smem(
                A_frag[(k_frag + 1) % 2][0], A_frag[(k_frag + 1) % 2][1],
                A_frag[(k_frag + 1) % 2][2], A_frag[(k_frag + 1) % 2][3],
                A_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float)
            );
            ptx::ld_smem(
                A_frag[(k_frag + 1) % 2][4], A_frag[(k_frag + 1) % 2][5],
                A_frag[(k_frag + 1) % 2][6], A_frag[(k_frag + 1) % 2][7],
                A_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float)
            );
            ptx::ld_smem(
                B_frag[(k_frag + 1) % 2][0], B_frag[(k_frag + 1) % 2][1],
                B_frag[(k_frag + 1) % 2][2], B_frag[(k_frag + 1) % 2][3],
                B_lds_addr + (k_frag + 1) % 8 * 128 * sizeof(float)
            );
            ptx::ld_smem(
                B_frag[(k_frag + 1) % 2][4], B_frag[(k_frag + 1) % 2][5],
                B_frag[(k_frag + 1) % 2][6], B_frag[(k_frag + 1) % 2][7],
                B_lds_addr + ((k_frag + 1) % 8 * 128 + 32) * sizeof(float)
            );
        }
        // 执行FFMA计算
        #pragma unroll
        for (uint32_t i = 0; i < 8; ++i) {
            #pragma unroll
            for (uint32_t j = 0; j < 8; ++j) {
                C_frag[i][j] += A_frag[k_frag % 2][i] * B_frag[k_frag % 2][j];
            }
        }
    }

    // 重用 128 * 48 * float 共享内存空间，行主序写回矩阵 C，每次写回一个分区的 16 * 32 * 8 = 128 * 32 * float 数据
    // [trid] = C_sts_addr + trid * 32 * sizeof(float);  trid = 0, 1, 2, 3, for 4x4 Thread Tile
    uint32_t C_sts_addr = ptx::smem_addr(smem_buf + warp_id * 16 * 32 + lane_rid * 4 * 32 + lane_cid * 4);
    // [iter] = C_lds_ptr + iter * 32;  iter = 0, 1, 2, ..., 15, for 16x32 Warp Tile
    const float *C_lds_ptr = reinterpret_cast<const float*>(smem_buf + warp_id * 16 * 32 + lane_id);
    // 将矩阵 C 写回设备内存时的，每个线程对应数据的偏移
    uint32_t m_idx = blockIdx.y * 128 + warp_id / 2 * 32;
    uint32_t n_idx = blockIdx.x * 128 + warp_id % 2 * 64 + lane_id;
    // [prid][pcid][iter] = C_stg_ptr + prid * 16 * N + pcid * 32 + iter * N
    float *C_stg_ptr = reinterpret_cast<float*>(C + m_idx * N + n_idx);
    #pragma unroll
    for (uint32_t prid = 0; prid < 2; ++prid) {
        #pragma unroll
        for (uint32_t pcid = 0; pcid < 2; ++pcid) {
            __syncthreads();
            #pragma unroll
            for (uint32_t trid = 0; trid < 4; ++trid) {
                ptx::st_smem(
                    C_frag[prid * 4 + trid][pcid * 4 + 0], C_frag[prid * 4 + trid][pcid * 4 + 1],
                    C_frag[prid * 4 + trid][pcid * 4 + 2], C_frag[prid * 4 + trid][pcid * 4 + 3],
                    C_sts_addr + trid * 32 * sizeof(float)
                );
            }
            __syncthreads();
            #pragma unroll
            for (uint32_t iter = 0; iter < 16; ++iter) {
                ptx::st_gmem(
                    C_lds_ptr[iter * 32], C_stg_ptr + prid * 16 * N + pcid * 32 + iter * N,
                    (m_idx + prid * 16 + iter < M) && (n_idx + pcid * 32 < N)
                );
            }
        }
    }
}

__device__ __forceinline__
void store_result_smem_rr(
    float Creg[8][8], float *smem_buf, float *C,
    const uint32_t M, const uint32_t N, const uint32_t cS,
    const uint32_t brid, const uint32_t bcid, const uint32_t tid,
    const uint32_t wrows, const uint32_t wcols, const uint32_t wrid, const uint32_t wcid,
    const uint32_t lrid, const uint32_t lcid
) {
    // 使用 32x128 共享内存搬运 128x128 数据（需 4 次），每次每线程写回 2x8 数据 Creg[r][:], Creg[r + 4][:]
    // [NEXT] C_smem_st + (tile_rid * wrows * 128 + tile_cid * wcols * 4) * sizeof(float)
    uint32_t C_smem_st = ptx::smem_addr(smem_buf + (wrid * wrows * 2 * 128 + wcid * wcols * 8) + (lrid * 128 + lcid * 4));
    float *C_block = C + (blockIdx.z * cS + brid * 128 * N + bcid * 128);
    for (uint32_t r = 0; r < 4; ++r) {
        __syncthreads();
        // 将数据写入到共享内存
        ptx::st_smem(Creg[r][0], Creg[r][1], Creg[r][2], Creg[r][3], C_smem_st + (0 * wrows * 128 + 0 * wcols * 4) * sizeof(float));
        ptx::st_smem(Creg[r][4], Creg[r][5], Creg[r][6], Creg[r][7], C_smem_st + (0 * wrows * 128 + 1 * wcols * 4) * sizeof(float));
        ptx::st_smem(Creg[r+4][0], Creg[r+4][1], Creg[r+4][2], Creg[r+4][3], C_smem_st + (1 * wrows * 128 + 0 * wcols * 4) * sizeof(float));
        ptx::st_smem(Creg[r+4][4], Creg[r+4][5], Creg[r+4][6], Creg[r+4][7], C_smem_st + (1 * wrows * 128 + 1 * wcols * 4) * sizeof(float));
        __syncthreads();
        // 使用 2x128 排列的线程搬运 32x128 共享内存（需 16 次），每次每线程写回 1 个数据
        #pragma unroll
        for (uint32_t gmem_row = r; gmem_row < 128; gmem_row += 4 * 2) {
            ptx::st_gmem(
                *reinterpret_cast<float*>(smem_buf + gmem_row / 4 * 128 + tid),
                C_block + (gmem_row + tid / 128 * 4) * N + (tid % 128),
                (brid * 128 + gmem_row + tid / 128 * 4 < M) && (bcid * 128 + tid % 128 < N)
            );
        }
    }
}

void sgemm_rrr(
    const float *A, const float *B, float *C,
    const uint32_t M, const uint32_t N, const uint32_t K
) {
    const dim3 block_size(256, 1, 1);
    const dim3 grid_size((N + 127) / 128, (M + 127) / 128, 1);
    sgemm_rrr_128x128x8_kernel<<<grid_size, block_size>>>(A, B, C, M, N, K);
}