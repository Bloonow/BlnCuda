#pragma once
#include <hip/hip_runtime.h>


#define time_it(__call, __repeat) do {                        \
    hipEvent_t __start, __end;                               \
    hipEventCreate(&__start);                                \
    hipEventCreate(&__end);                                  \
    hipEventRecord(__start);                                 \
    hipEventQuery(__start);                                  \
    (__call);                                                 \
    for (int __i = 0; __i < __repeat; __i++) (__call);        \
    hipEventRecord(__end);                                   \
    hipEventSynchronize(__end);                              \
    float __elapse;                                           \
    hipEventElapsedTime(&__elapse, __start, __end);          \
    hipEventDestroy(__start);                                \
    hipEventDestroy(__end);                                  \
    printf("Elapse = %g ms, Average = %g ms, Repeat = %d.\n", \
            __elapse, __elapse / __repeat, __repeat);         \
} while (0);
// #define time_it(__call, __repeat)

#define time_it_v2(__call, __repeat, __ave) do {              \
    hipEvent_t __start, __end;                               \
    hipEventCreate(&__start);                                \
    hipEventCreate(&__end);                                  \
    hipEventRecord(__start);                                 \
    hipEventQuery(__start);                                  \
    (__call);                                                 \
    for (int __i = 0; __i < __repeat; __i++) (__call);        \
    hipEventRecord(__end);                                   \
    hipEventSynchronize(__end);                              \
    float __elapse;                                           \
    hipEventElapsedTime(&__elapse, __start, __end);          \
    __ave = __elapse / __repeat;                              \
    hipEventDestroy(__start);                                \
    hipEventDestroy(__end);                                  \
} while (0);
// #define time_it_v2(__call, __repeat, __ave)