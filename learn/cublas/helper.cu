/**
 * All vectors and matrices store in column-major format.
 * 🤣 👉 🤡
 * Matrices can store both in row-major and column-major now.
*/

#include <hip/hip_runtime.h>
#include <time.h>
#include <stdarg.h>

template<typename Ty>
Ty* alloc_host_memory(size_t count, Ty max_init = (Ty)(1)) {
    static time_t init_time = 0;
    if (init_time == 0) srand(time(&init_time));
    Ty* ptr = reinterpret_cast<Ty*>(malloc(sizeof(Ty) * count));
    for (size_t i = 0; i < count; i++) {
        ptr[i] = static_cast<Ty>(static_cast<double>(rand()) / RAND_MAX * max_init);
    }
    return ptr;
}

template<typename Ty>
Ty* alloc_cuda_memory(size_t count, Ty *host_ptr = nullptr) {
    Ty* ptr;
    hipMalloc(&ptr, sizeof(Ty) * count);
    if (host_ptr != nullptr) {
        hipMemcpy(ptr, host_ptr, sizeof(Ty) * count, hipMemcpyHostToDevice);
    }
    return ptr;
}

void free_memory(size_t count, ...) {
    va_list vlist;
    va_start(vlist, count);
    for (size_t i = 0; i < count; i++) {
        void *ptr = va_arg(vlist, void*);
        // try free as gpu pointer
        hipError_t status = hipFree(ptr);
        // free as host pointer
        if (status != 0) free(ptr);
    }
    va_end(vlist);
}

template<typename Ty>
bool check_same(Ty *ptr1, Ty *ptr2, size_t num, Ty error = 1e-4) {
    for (size_t i = 0; i < num; i++) {
        if (abs(ptr1[i] - ptr2[i]) > error) return false;
    }
    return true;
}

typedef enum {
    ROW_MAJOR = 0,
    COL_MAJOR = 1
} StorageOrder_t;

inline size_t row_index(size_t rid, size_t cid, size_t rows, size_t cols) {
    return rid * cols + cid;
}

inline size_t col_index(size_t rid, size_t cid, size_t rows, size_t cols) {
    return cid * rows + rid;
}

template<typename Ty>
void host_gemv(
    size_t M, size_t N, StorageOrder_t A_order, Ty *A, Ty *x, Ty *y, Ty alpha, Ty beta, size_t batch_count
) {
    auto A_index = A_order == ROW_MAJOR ? row_index : col_index;
    for (size_t bid = 0; bid < batch_count; bid++) {
        for (size_t rid = 0; rid < M; rid++) {
            Ty value = 0;
            for (size_t cid = 0; cid < N; cid++) {
                value += A[bid * M * N + A_index(rid, cid, M, N)] * x[bid * N + cid];
            }
            y[bid * M + rid] = alpha * value + beta * y[bid * M + rid];
        }
    }
}
