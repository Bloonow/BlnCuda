#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../../utils/helper.cu"

int main(int argc, char *argv[]) {
    size_t Batch = 4, M = 456, N = 987;
    float alpha = 3.14, beta = 2.71;
    float *h_A = alloc_host_memory<float>(Batch * M * N);
    float *h_x = alloc_host_memory<float>(Batch * N);
    float *h_y = alloc_host_memory<float>(Batch * M);
    float *ret_y = alloc_host_memory<float>(Batch * M);
    float *d_A = alloc_cuda_memory<float>(Batch * M * N, h_A);
    float *d_x = alloc_cuda_memory<float>(Batch * N, h_x);
    float *d_y = alloc_cuda_memory<float>(Batch * M, h_y);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemvStridedBatched(
        handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, M * N, d_x, 1, N, &beta, d_y, 1, M, Batch
    );
    hipMemcpy(ret_y, d_y, sizeof(float) * Batch * M, hipMemcpyDeviceToHost);
    hipblasDestroy(handle);

    host_gemv<float, col_major>(h_A, h_x, h_y, alpha, beta, M, N, Batch);
    check_same<float>(h_y, ret_y, Batch * M, 1e-3);

    free_memory(7, h_A, h_x, h_y, ret_y, d_A, d_x, d_y);
    return 0;
}