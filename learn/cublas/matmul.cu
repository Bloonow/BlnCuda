#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include "../helper.cu"

int main(int argc, char *argv[]) {
    int32_t Batch = 4;
    int64_t M = 456, N = 987, K = 543;
    uint64_t workspace_bytes = 16 * 1024 * 1024;
    float alpha = 3.14, beta = 2.71;
    float *h_A = alloc_host_memory<float>(Batch * M * K);
    float *h_B = alloc_host_memory<float>(Batch * K * N);
    float *h_C = alloc_host_memory<float>(Batch * M * N);
    float *h_D = alloc_host_memory<float>(Batch * M * N);
    float *h_bias = alloc_host_memory<float>(Batch * M);
    float *ret_D = alloc_host_memory<float>(Batch * M * N);
    float *d_A = alloc_cuda_memory<float>(Batch * M * K, h_A);
    float *d_B = alloc_cuda_memory<float>(Batch * K * N, h_B);
    float *d_C = alloc_cuda_memory<float>(Batch * M * N, h_C);
    float *d_D = alloc_cuda_memory<float>(Batch * M * N, h_D);
    float *d_bias = alloc_cuda_memory<float>(Batch * M, h_bias);
    int64_t bitmask_ld = (M + 127) / 128 * 128;
    char *bitmask = alloc_cuda_memory<char>(Batch * N * bitmask_ld / (8 * sizeof(char)), nullptr);
    void *workspace = alloc_cuda_memory<char>(workspace_bytes / sizeof(char), nullptr);

    hipblasLtHandle_t lt;
    hipblasStatus_t stat = hipblasLtCreate(&lt);

    // 设置矩阵布局
    hipblasLtOrder_t order = HIPBLASLT_ORDER_ROW;
    hipblasLtMatrixLayout_t A_layout, B_layout, C_layout, D_layout;
    stat = hipblasLtMatrixLayoutCreate(&A_layout, HIP_R_32F, M, K, order == HIPBLASLT_ORDER_COL ? M : K);
    stat = hipblasLtMatrixLayoutCreate(&B_layout, HIP_R_32F, K, N, order == HIPBLASLT_ORDER_COL ? K : N);
    stat = hipblasLtMatrixLayoutCreate(&C_layout, HIP_R_32F, M, N, order == HIPBLASLT_ORDER_COL ? M : N);
    stat = hipblasLtMatrixLayoutCreate(&D_layout, HIP_R_32F, M, N, order == HIPBLASLT_ORDER_COL ? M : N);
    // 设置行主序存储或列主序存储
    stat = hipblasLtMatrixLayoutSetAttribute(A_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
    stat = hipblasLtMatrixLayoutSetAttribute(B_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
    stat = hipblasLtMatrixLayoutSetAttribute(C_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
    stat = hipblasLtMatrixLayoutSetAttribute(D_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
    // 设置批量数目及跨步间距
    stat = hipblasLtMatrixLayoutSetAttribute(A_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(Batch));
    stat = hipblasLtMatrixLayoutSetAttribute(B_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(Batch));
    stat = hipblasLtMatrixLayoutSetAttribute(C_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(Batch));
    stat = hipblasLtMatrixLayoutSetAttribute(D_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(Batch));
    int64_t A_stride = M * K, B_stride = K * N, C_stride = M * N, D_stride = M * N;
    stat = hipblasLtMatrixLayoutSetAttribute(A_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &A_stride, sizeof(A_stride));
    stat = hipblasLtMatrixLayoutSetAttribute(B_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &B_stride, sizeof(B_stride));
    stat = hipblasLtMatrixLayoutSetAttribute(C_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &C_stride, sizeof(C_stride));
    stat = hipblasLtMatrixLayoutSetAttribute(D_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &D_stride, sizeof(D_stride));

    // 设置矩阵乘法计算配置
    hipblasLtMatmulDesc_t matmul_desc;
    stat = hipblasLtMatmulDescCreate(&matmul_desc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
    // 设置矩阵乘法后置操作
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_RELU_AUX_BIAS;
    int64_t bitmask_stride = bitmask_ld * N;
    stat = hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
    stat = hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &d_bias, sizeof(d_bias));
    stat = hipblasLtMatmulDescSetAttribute(matmul_desc, CUBLASLT_MATMUL_DESC_BIAS_BATCH_STRIDE, &M, sizeof(M));
    stat = hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &bitmask, sizeof(bitmask));
    stat = hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &bitmask_ld, sizeof(bitmask_ld));
    stat = hipblasLtMatmulDescSetAttribute(
        matmul_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_BATCH_STRIDE, &bitmask_stride, sizeof(bitmask_stride)
    );

    // 搜索最佳的实现算法
    const int requestAlgoCount = 4;
    int returnAlgoCount = 0;
    hipblasLtMatmulPreference_t preference;
    stat = hipblasLtMatmulPreferenceCreate(&preference);
    stat = hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspace_bytes, sizeof(workspace_bytes)
    );
    hipblasLtMatmulHeuristicResult_t heuristicResult[requestAlgoCount] = { 0 };
    stat = hipblasLtMatmulAlgoGetHeuristic(
        lt, matmul_desc, A_layout, B_layout, C_layout, D_layout, preference,
        requestAlgoCount, heuristicResult, &returnAlgoCount
    );

    // 矩阵乘法
    stat = hipblasLtMatmul(
        lt, matmul_desc, &alpha, d_A, A_layout, d_B, B_layout, &beta, d_C, C_layout, d_D, D_layout,
        &heuristicResult[0].algo, workspace, workspace_bytes, nullptr
    );
    hipMemcpy(ret_D, d_D, sizeof(float) * Batch * M * N, hipMemcpyDeviceToHost);

    // 主机乘法验证

    // 释放资源
    hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatmulDescDestroy(matmul_desc);
    hipblasLtMatrixLayoutDestroy(D_layout);
    hipblasLtMatrixLayoutDestroy(C_layout);
    hipblasLtMatrixLayoutDestroy(B_layout);
    hipblasLtMatrixLayoutDestroy(A_layout);
    hipblasLtDestroy(lt);
    free_memory(13, h_A, h_B, h_C, h_D, h_bias, ret_D, d_A, d_B, d_C, d_D, d_bias, bitmask, workspace);
    return 0;
}