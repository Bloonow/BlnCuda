#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include "../../utils/helper.cu"

int main(int argc, char *argv[]) {
    int32_t Batch = 4;
    int64_t M = 456, N = 987, K = 543;
    uint64_t workspace_bytes = 16 * 1024 * 1024;
    float alpha = 3.14, beta = 2.71;
    float *h_A = alloc_host_memory<float>(Batch * M * K);
    float *h_B = alloc_host_memory<float>(Batch * K * N);
    float *h_C = alloc_host_memory<float>(Batch * M * N);
    float *h_D = alloc_host_memory<float>(Batch * M * N);
    float *h_bias = alloc_host_memory<float>(Batch * M);
    float *ret_D = alloc_host_memory<float>(Batch * M * N);
    float *d_A = alloc_cuda_memory<float>(Batch * M * K, h_A);
    float *d_B = alloc_cuda_memory<float>(Batch * K * N, h_B);
    float *d_C = alloc_cuda_memory<float>(Batch * M * N, h_C);
    float *d_D = alloc_cuda_memory<float>(Batch * M * N, h_D);
    float *d_bias = alloc_cuda_memory<float>(Batch * M, h_bias);
    int64_t bitmask_ld = (M + 127) / 128 * 128;
    char *bitmask = alloc_cuda_memory<char>(Batch * N * bitmask_ld / (8 * sizeof(char)), nullptr);
    void *workspace = alloc_cuda_memory<char>(workspace_bytes / sizeof(char), nullptr);

    hipblasLtHandle_t lt;
    hipblasLtCreate(&lt);

    // 设置矩阵布局
    hipblasLtOrder_t order = HIPBLASLT_ORDER_COL;
    hipblasLtMatrixLayout_t A_layout, B_layout, C_layout, D_layout;
    hipblasLtMatrixLayoutCreate(&A_layout, HIP_R_32F, M, K, order == HIPBLASLT_ORDER_COL ? M : K);
    hipblasLtMatrixLayoutCreate(&B_layout, HIP_R_32F, K, N, order == HIPBLASLT_ORDER_COL ? K : N);
    hipblasLtMatrixLayoutCreate(&C_layout, HIP_R_32F, M, N, order == HIPBLASLT_ORDER_COL ? M : N);
    hipblasLtMatrixLayoutCreate(&D_layout, HIP_R_32F, M, N, order == HIPBLASLT_ORDER_COL ? M : N);
    // 设置行主序存储或列主序存储
    hipblasLtMatrixLayoutSetAttribute(A_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(hipblasLtOrder_t));
    hipblasLtMatrixLayoutSetAttribute(B_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(hipblasLtOrder_t));
    hipblasLtMatrixLayoutSetAttribute(C_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(hipblasLtOrder_t));
    hipblasLtMatrixLayoutSetAttribute(D_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(hipblasLtOrder_t));
    // 设置批量数目及跨步间距
    hipblasLtMatrixLayoutSetAttribute(A_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(int32_t));
    hipblasLtMatrixLayoutSetAttribute(B_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(int32_t));
    hipblasLtMatrixLayoutSetAttribute(C_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(int32_t));
    hipblasLtMatrixLayoutSetAttribute(D_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(int32_t));
    int64_t A_stride = M * K, B_stride = K * N, C_stride = M * N, D_stride = M * N;
    hipblasLtMatrixLayoutSetAttribute(A_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &A_stride, sizeof(int64_t));
    hipblasLtMatrixLayoutSetAttribute(B_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &B_stride, sizeof(int64_t));
    hipblasLtMatrixLayoutSetAttribute(C_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &C_stride, sizeof(int64_t));
    hipblasLtMatrixLayoutSetAttribute(D_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &D_stride, sizeof(int64_t));

    // 设置矩阵乘法计算配置
    hipblasLtMatmulDesc_t matmul_desc;
    hipblasLtMatmulDescCreate(&matmul_desc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
    // 设置矩阵乘法后置操作
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_RELU_AUX_BIAS;
    int64_t bitmask_stride = bitmask_ld * N;
    hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(hipblasLtEpilogue_t));
    hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &d_bias, sizeof(const void*));
    hipblasLtMatmulDescSetAttribute(matmul_desc, CUBLASLT_MATMUL_DESC_BIAS_BATCH_STRIDE, &M, sizeof(int64_t));
    hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &bitmask, sizeof(void*));
    hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &bitmask_ld, sizeof(int64_t));
    hipblasLtMatmulDescSetAttribute(
        matmul_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_BATCH_STRIDE, &bitmask_stride, sizeof(int64_t)
    );

    // 搜索最佳的实现算法
    const int requestAlgoCount = 4;
    int returnAlgoCount = 0;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatmulPreferenceCreate(&preference);
    hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspace_bytes, sizeof(uint64_t)
    );
    hipblasLtMatmulHeuristicResult_t heuristicResult[requestAlgoCount] = { 0 };
    hipblasLtMatmulAlgoGetHeuristic(
        lt, matmul_desc, A_layout, B_layout, C_layout, D_layout, preference,
        requestAlgoCount, heuristicResult, &returnAlgoCount
    );

    // 矩阵乘法
    hipblasLtMatmul(
        lt, matmul_desc, &alpha, d_A, A_layout, d_B, B_layout, &beta, d_C, C_layout, d_D, D_layout,
        &heuristicResult[0].algo, workspace, workspace_bytes, nullptr
    );
    hipMemcpy(ret_D, d_D, sizeof(float) * Batch * M * N, hipMemcpyDeviceToHost);

    // 主机乘法验证
    host_matmul_relu<float, col_major, col_major, col_major, col_major>(
        h_A, h_B, h_C, h_D, h_bias, alpha, beta, M, N, K, Batch
    );
    check_same<float>(h_D, ret_D, Batch * M * N, 1e-4);

    // 释放资源
    hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatmulDescDestroy(matmul_desc);
    hipblasLtMatrixLayoutDestroy(D_layout);
    hipblasLtMatrixLayoutDestroy(C_layout);
    hipblasLtMatrixLayoutDestroy(B_layout);
    hipblasLtMatrixLayoutDestroy(A_layout);
    hipblasLtDestroy(lt);
    free_memory(13, h_A, h_B, h_C, h_D, h_bias, ret_D, d_A, d_B, d_C, d_D, d_bias, bitmask, workspace);
    return 0;
}