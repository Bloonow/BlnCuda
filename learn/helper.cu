/**
 * All vectors and matrices store in column-major format.
 * 🤣 👉 🤡
 * Matrices can store both in row-major and column-major now.
*/

#include <hip/hip_runtime.h>
#include <time.h>
#include <stdarg.h>

template<typename Ty>
Ty* alloc_host_memory_increment(size_t count, Ty start = (Ty)(0)) {
    Ty *ptr = reinterpret_cast<Ty*>(malloc(sizeof(Ty) * count));
    Ty value = start;
    for (size_t i = 0; i < count; ptr[i++] = value++);
    return ptr;
}

template<typename Ty>
Ty* alloc_host_memory(size_t count, Ty max_init = (Ty)(1)) {
    static time_t init_time = 0;
    if (init_time == 0) srand(time(&init_time));
    Ty *ptr = reinterpret_cast<Ty*>(malloc(sizeof(Ty) * count));
    for (size_t i = 0; i < count; i++) {
        Ty value = rand() % 2 == 0 ? 1 : -1;
        value *= static_cast<Ty>(static_cast<double>(rand()) / RAND_MAX * max_init);
        ptr[i] = value;
    }
    return ptr;
}

template<typename Ty>
Ty* alloc_cuda_memory(size_t count, Ty *host_ptr = nullptr) {
    Ty *ptr;
    hipMalloc(&ptr, sizeof(Ty) * count);
    if (host_ptr != nullptr) {
        hipMemcpy(ptr, host_ptr, sizeof(Ty) * count, hipMemcpyHostToDevice);
    }
    return ptr;
}

void free_memory(size_t count, ...) {
    va_list vlist;
    va_start(vlist, count);
    for (size_t i = 0; i < count; i++) {
        void *ptr = va_arg(vlist, void*);
        // free as cuda pointer
        hipError_t status = hipFree(ptr);
        // free as host pointer
        if (status != hipSuccess) free(ptr);
    }
    va_end(vlist);
}

template<typename Ty>
bool check_same(Ty *ptr1, Ty *ptr2, size_t num, Ty error = 1e-6) {
    for (size_t i = 0; i < num; i++) {
        if (abs(ptr1[i] - ptr2[i]) > error) {
            printf("[%ld], p1 = %f, p2 = %f, diff = %f\n", i, ptr1[i], ptr2[i], abs(ptr1[i] - ptr2[i]));
            return false;
        }
    }
    return true;
}

typedef enum {
    ROW_MAJOR = 0,
    COL_MAJOR = 1
} Order_t;

inline size_t row_index(size_t bid, size_t rid, size_t cid, size_t rows, size_t cols) {
    return bid * rows * cols + rid * cols + cid;
}

inline size_t col_index(size_t bid, size_t rid, size_t cid, size_t rows, size_t cols) {
    return bid * rows * cols + cid * rows + rid;
}

template<typename Ty>
void host_gemv(
    size_t M, size_t N, Order_t A_order,
    Ty *A, Ty *x, Ty *y, Ty alpha, Ty beta, size_t batch_count
) {
    auto A_idx = A_order == ROW_MAJOR ? row_index : col_index;
    for (size_t bid = 0; bid < batch_count; bid++) {
        for (size_t rid = 0; rid < M; rid++) {
            Ty value = 0;
            for (size_t cid = 0; cid < N; cid++) {
                value += A[A_idx(bid, rid, cid, M, N)] * x[bid * N + cid];
            }
            y[bid * M + rid] = alpha * value + beta * y[bid * M + rid];
        }
    }
}

template<typename Ty>
void host_gemm(
    size_t M, size_t N, size_t K, Order_t A_order, Order_t B_order, Order_t C_order,
    Ty *A, Ty *B, Ty *C, Ty alpha, Ty beta, size_t batch_count
) {
    auto A_idx = A_order == ROW_MAJOR ? row_index : col_index;
    auto B_idx = B_order == ROW_MAJOR ? row_index : col_index;
    auto C_idx = C_order == ROW_MAJOR ? row_index : col_index;
    for (size_t bid = 0; bid < batch_count; bid++) {
        for (size_t rid = 0; rid < M; rid++) {
            for (size_t cid = 0; cid < N; cid++) {
                Ty value = 0;
                for (size_t k = 0; k < K; k++) {
                    value += A[A_idx(bid, rid, k, M, K)] * B[B_idx(bid, k, cid, K, N)];
                }
                C[C_idx(bid, rid, cid, M, N)] = alpha * value + beta * C[C_idx(bid, rid, cid, M, N)];
            }
        }
    }
}

template<typename Ty>
void host_matmul_relu(
    size_t M, size_t N, size_t K, Order_t A_order, Order_t B_order, Order_t C_order, Order_t D_order,
    Ty *A, Ty *B, Ty *C, Ty* D, Ty *bias, Ty alpha, Ty beta, size_t batch_count
) {
    auto A_idx = A_order == ROW_MAJOR ? row_index : col_index;
    auto B_idx = B_order == ROW_MAJOR ? row_index : col_index;
    auto C_idx = C_order == ROW_MAJOR ? row_index : col_index;
    auto D_idx = D_order == ROW_MAJOR ? row_index : col_index;
    for (size_t bid = 0; bid < batch_count; bid++) {
        for (size_t rid = 0; rid < M; rid++) {
            for (size_t cid = 0; cid < N; cid++) {
                Ty value = 0;
                for (size_t k = 0; k < K; k++) {
                    value += A[A_idx(bid, rid, k, M, K)] * B[B_idx(bid, k, cid, K, N)];
                }
                value = alpha * value + beta * C[C_idx(bid, rid, cid, M, N)];
                value = value + bias[bid * M + rid];
                value = value > 0 ? value : 0;
                D[D_idx(bid, rid, cid, M, N)] = value;
            }
        }
    }
}