#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

__global__ void scale_kernel(hipfftComplex* data, float factor, const int count) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < count) {
        data[tid].x *= factor;
        data[tid].y *= factor;
    }
}

int main(int argc, char *argv[]) {
    int batch = 8;
    int N1 = 256, N2 = 128;
    hipfftReal *h_input = (hipfftReal*)malloc(batch * N1 * N2 * sizeof(hipfftReal));
    hipfftReal *h_reslut = (hipfftReal*)malloc(batch * N1 * N2 * sizeof(hipfftReal));
    for (int i = 0; i < batch * N1 * N2; i++) h_input[i] = i * 1.f / batch;
    hipfftReal *input, *result;
    hipfftComplex *intermediate;
    hipMalloc((hipfftReal**)(&input), batch * N1 * N2 * sizeof(hipfftReal));
    hipMalloc((hipfftReal**)(&result), batch * N1 * N2 * sizeof(hipfftReal));
    hipMalloc((hipfftComplex**)(&intermediate), batch * N1 * (N2 / 2 + 1) * sizeof(hipfftComplex));
    hipMemcpy(input, h_input, batch * N1 * N2 * sizeof(hipfftReal), hipMemcpyHostToDevice);

    int N[2] = {N1, N2};
    hipfftHandle plan2D_r2c, plan2D_c2r;
    hipfftCreate(&plan2D_r2c);
    hipfftCreate(&plan2D_c2r);
    // 构建plan配置
    hipfftPlanMany(&plan2D_r2c, 2, N, nullptr, 1, N1 * N2, nullptr, 1, N1 * (N2 / 2 + 1), HIPFFT_R2C, batch);
    hipfftPlanMany(&plan2D_c2r, 2, N, nullptr, 1, N1 * (N2 / 2 + 1), nullptr, 1, N1 * N2, HIPFFT_C2R, batch);
    hipfftExecR2C(plan2D_r2c, input, intermediate);   // 正变换
    // 因为傅里叶逆变换需要除以N，故在变换之前先进行标准化，也可以在变换之后进行标准化
    scale_kernel<<<(batch * N1 * (N2 / 2 + 1) + 127) / 128, 128>>>(
        intermediate, 1.f / (N1 * N2), batch * N1 * (N2 / 2 + 1)
    );
    hipfftExecC2R(plan2D_c2r, intermediate, result);  // 逆变换
    hipfftDestroy(plan2D_r2c);
    hipfftDestroy(plan2D_c2r);

    bool all_same = true;
    hipMemcpy(h_reslut, result, batch * N1 * N2 * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    for (int i = 0; i < batch * N1 * N2; i++) {
        if (abs(h_input[i] - h_reslut[i]) > 1.e-3) {
            all_same = false;
            break;
        }
    }
    printf("The data are %s.\n", all_same ? "all same" : "not all same");

    hipFree(input);
    hipFree(result);
    hipFree(intermediate);
    free(h_input);
    free(h_reslut);
    return 0;
}