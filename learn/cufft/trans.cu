#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "../../utils/helper.cu"

__global__ void scale_kernel(hipfftComplex* data, float factor, const int count) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < count) {
        data[tid].x *= factor;
        data[tid].y *= factor;
    }
}

int main(int argc, char *argv[]) {
    int Batch = 8;
    int N1 = 256, N2 = 128;
    hipfftReal *h_input = (hipfftReal*)malloc(Batch * N1 * N2 * sizeof(hipfftReal));
    hipfftReal *h_reslut = (hipfftReal*)malloc(Batch * N1 * N2 * sizeof(hipfftReal));
    for (int i = 0; i < Batch * N1 * N2; i++) h_input[i] = i * 1.f / Batch;
    hipfftReal *d_input, *d_result;
    hipfftComplex *d_temp;
    hipMalloc((hipfftReal**)(&d_input), Batch * N1 * N2 * sizeof(hipfftReal));
    hipMalloc((hipfftReal**)(&d_result), Batch * N1 * N2 * sizeof(hipfftReal));
    hipMalloc((hipfftComplex**)(&d_temp), Batch * N1 * (N2 / 2 + 1) * sizeof(hipfftComplex));
    hipMemcpy(d_input, h_input, Batch * N1 * N2 * sizeof(hipfftReal), hipMemcpyHostToDevice);

    hipfftHandle plan2D_r2c, plan2D_c2r;
    hipfftCreate(&plan2D_r2c);
    hipfftCreate(&plan2D_c2r);
    // 构建plan配置
    int Ranks[2] = { N1, N2 };
    int R_nembed[2] = { N1, N2 };
    int C_nembed[2] = { N1, N2 / 2 + 1 };
    hipfftPlanMany(&plan2D_r2c, 2, Ranks, R_nembed, 1, N1 * N2, C_nembed, 1, N1 * (N2 / 2 + 1), HIPFFT_R2C, Batch);
    hipfftPlanMany(&plan2D_c2r, 2, Ranks, C_nembed, 1, N1 * (N2 / 2 + 1), R_nembed, 1, N1 * N2, HIPFFT_C2R, Batch);
    // 正变换
    hipfftExecR2C(plan2D_r2c, d_input, d_temp);
    // 标准化，在逆变换之前进行
    scale_kernel<<<(Batch * N1 * (N2 / 2 + 1) + 127) / 128, 128>>>(d_temp, 1.f / (N1 * N2), Batch * N1 * (N2 / 2 + 1));
    // 逆变换
    hipfftExecC2R(plan2D_c2r, d_temp, d_result);
    hipfftDestroy(plan2D_r2c);
    hipfftDestroy(plan2D_c2r);

    hipMemcpy(h_reslut, d_result, Batch * N1 * N2 * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    check_same<hipfftReal>(h_reslut, h_input, Batch * N1 * N2, 1.e-3);

    free_memory(5, h_input, h_reslut, d_input, d_result, d_temp);
    return 0;
}