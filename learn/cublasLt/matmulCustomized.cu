#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>

int main(int argc, char *argv[]) {
    int M = 2;
    int K = 3;
    int N = 2;
    int32_t batch = 2;
    float *h_A = (float*)malloc(sizeof(float) * M * K * batch);  // X, input
    float *h_B = (float*)malloc(sizeof(float) * K * N);          // W, weight
    float *h_C = (float*)malloc(sizeof(float) * 1 * N);          // B, bias
    float *h_D = (float*)malloc(sizeof(float) * M * N * batch);  // Y = X W + B
    for (int i = 0; i < M * K * batch; i++) h_A[i] = (i % (M * K)) * 1.0;
    for (int i = 0; i < K * N; i++)         h_B[i] = i * 1.0;
    for (int i = 0; i < 1 * N; i++)         h_C[i] = i * 0.1;
    for (int i = 0; i < M * N * batch; i++) h_D[i] = 0.0;
    printf("A\t=\t");
    for (int i = 0; i < M * K * batch; i++) printf("%.1f\t", h_A[i]);
    printf("\n");
    printf("B\t=\t");
    for (int i = 0; i < K * N; i++) printf("%.1f\t", h_B[i]);
    printf("\n");
    printf("C\t=\t");
    for (int i = 0; i < 1 * N; i++) printf("%.1f\t", h_C[i]);
    printf("\n");
    
    float *A, *B, *C, *D;
    hipMalloc(&A, sizeof(float) * M * K * batch);
    hipMalloc(&B, sizeof(float) * K * N);
    hipMalloc(&C, sizeof(float) * 1 * N);
    hipMalloc(&D, sizeof(float) * M * N * batch);
    hipMemcpy(A, h_A, sizeof(float) * M * K * batch, hipMemcpyHostToDevice);
    hipMemcpy(B, h_B, sizeof(float) * K * N, hipMemcpyHostToDevice);
    hipMemcpy(C, h_C, sizeof(float) * 1 * N, hipMemcpyHostToDevice);

    hipblasLtHandle_t handle;
    hipblasLtCreate(&handle);
    hipblasLtMatmulDesc_t computeDesc;
    hipblasLtMatmulDescCreate(&computeDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc, Ddesc;  // row-major
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, M, K, K);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, K, N, N);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, M, N, 0);  // broadcast
    hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_32F, M, N, N);
    hipblasLtOrder_t row_major = HIPBLASLT_ORDER_ROW;
    hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &row_major, sizeof(hipblasLtOrder_t));
    hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &row_major, sizeof(hipblasLtOrder_t));
    hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &row_major, sizeof(hipblasLtOrder_t));
    hipblasLtMatrixLayoutSetAttribute(
        Ddesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &row_major, sizeof(hipblasLtOrder_t));
    hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(int32_t));
    hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(int32_t));
    hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(int32_t));
    hipblasLtMatrixLayoutSetAttribute(
        Ddesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch, sizeof(int32_t));
    int64_t Astride = M * K;
    int64_t Bstride = 0;
    int64_t Cstride = 0;
    int64_t Dstride = M * N;
    hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &Astride, sizeof(int64_t));
    hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &Bstride, sizeof(int64_t));
    hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &Cstride, sizeof(int64_t));
    hipblasLtMatrixLayoutSetAttribute(
        Ddesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &Dstride, sizeof(int64_t));

    float alpha = 1.0, beta = 1.0;
    // Not use workspace
    hipblasLtMatmul(
        handle, computeDesc, &alpha, A, Adesc, B, Bdesc, &beta,
        C, Cdesc, D, Ddesc, NULL, NULL, 0, NULL
    );

    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
    hipblasLtMatrixLayoutDestroy(Ddesc);
    hipblasLtMatmulDescDestroy(computeDesc);
    hipblasLtDestroy(handle);

    hipMemcpy(h_D, D, sizeof(float) * M * N * batch, hipMemcpyDeviceToHost);
    printf("D\t=\t"); 
    for (int i = 0; i < M * N * batch; i++) printf("%.1f\t", h_D[i]); 
    printf("\n");
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(D);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);
    return 0;
}