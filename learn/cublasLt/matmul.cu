#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>

int main(int argc, char *argv[]) {
    int M = 2;
    int K = 3;
    int N = 4;

    float *h_A = (float*)malloc(sizeof(float) * M * K);
    float *h_B = (float*)malloc(sizeof(float) * K * N);
    float *h_C = (float*)malloc(sizeof(float) * M * N);
    float *h_D = (float*)malloc(sizeof(float) * M * N);
    for (int i = 0; i < M * K; i++) h_A[i] = i * 1.0;
    for (int i = 0; i < K * N; i++) h_B[i] = i * 1.0;
    for (int i = 0; i < M * N; i++) h_C[i] = i * 1.0;
    for (int i = 0; i < M * N; i++) h_D[i] = 0.0;
    printf("A\t=\t"); for (int i = 0; i < M * K; i++) printf("%.1f\t", h_A[i]); printf("\n");
    printf("B\t=\t"); for (int i = 0; i < K * N; i++) printf("%.1f\t", h_B[i]); printf("\n");
    printf("C\t=\t"); for (int i = 0; i < M * N; i++) printf("%.1f\t", h_C[i]); printf("\n");
    
    float *A, *B, *C, *D;
    hipMalloc(&A, sizeof(float) * M * K);
    hipMalloc(&B, sizeof(float) * K * N);
    hipMalloc(&C, sizeof(float) * M * N);
    hipMalloc(&D, sizeof(float) * M * N);
    hipMemcpy(A, h_A, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(B, h_B, sizeof(float) * K * N, hipMemcpyHostToDevice);
    hipMemcpy(C, h_C, sizeof(float) * M * N, hipMemcpyHostToDevice);

    hipblasLtHandle_t handle;
    hipblasLtCreate(&handle);
    hipblasLtMatmulDesc_t computeDesc;
    hipblasLtMatmulDescCreate(&computeDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc, Ddesc;  // D = AB + C
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, M, K, M);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, K, N, K);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, M, N, M);
    hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_32F, M, N, M);

    float alpha = 1.0, beta = 1.0;
    // A, B, C, D are stored as column-major  // Not use workspace
    hipblasLtMatmul(
        handle, computeDesc, &alpha, A, Adesc, B, Bdesc, &beta, 
        C, Cdesc, D, Ddesc, NULL, NULL, 0, NULL
    );
    
    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
    hipblasLtMatrixLayoutDestroy(Ddesc);
    hipblasLtMatmulDescDestroy(computeDesc);
    hipblasLtDestroy(handle);

    hipMemcpy(h_D, D, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    printf("D\t=\t"); for (int i = 0; i < M * N; i++) printf("%.1f\t", h_D[i]); printf("\n");
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(D);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);
    return 0;
}