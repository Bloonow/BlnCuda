#include "../utils/helper.cu"
#include "../gemm/hgemm_128x128.cu"
#include "../gemm/gemm.cu"

int main(int argc, char *argv[]) {
    int M = 5120 * 1, N = 5120 * 1, K = 2560 * 1;
    float alpha = 1.5, beta = 2.78;
    half* h_A = alloc_host_memory<half>(M * K);
    half* h_B = alloc_host_memory<half>(K * N);
    float* h_C = alloc_host_memory<float>(M * N);
    float *ret_D0 = alloc_host_memory<float>(M * N);
    float *ret_D1 = alloc_host_memory<float>(M * N);
    half* d_A = alloc_cuda_memory<half>(M * K, h_A);
    half* d_B = alloc_cuda_memory<half>(K * N, h_B);
    float* d_C = alloc_cuda_memory<float>(M * N, h_C);
    float* d_D = alloc_cuda_memory<float>(M * N);

    wmma_hgemm_m16n16k16::wmma_hgemm_rcr_cuda(d_A, d_B, d_C, d_D, alpha, beta, M, N, K);
    hipMemcpy(ret_D0, d_D, M * N * sizeof(float), hipMemcpyDeviceToHost);

    cublasLt_hgemm(d_A, d_B, d_C, alpha, beta, M, N, K, 1, HIPBLASLT_ORDER_ROW, HIPBLASLT_ORDER_COL, HIPBLASLT_ORDER_ROW);
    hipMemcpy(ret_D1, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // host_gemm<row_major, col_major, row_major>(h_A, h_B, h_C, 1, 0, M, N, K, 1);
    check_same<float>(ret_D0, ret_D1, M * N, 1.e-3);

    free_memory(9, h_A, h_B, h_C, ret_D0, ret_D1, d_A, d_B, d_C, d_D);
    return 0;
}