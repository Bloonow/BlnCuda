#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include "../learn/helper.cu"

int main(int argc, char *argv[]) {
    size_t M = 64, N = 64;
    int8_t alpha = 1, beta = 0;
    int8_t *h1 = alloc_host_memory_increment<int8_t>(M * N);
    int8_t *h2 = alloc_host_memory<int8_t>(M * N);
    int8_t *d1 = alloc_cuda_memory<int8_t>(M * N, h1);
    int8_t *d2 = alloc_cuda_memory<int8_t>(M * N, nullptr);

    hipblasStatus_t stat;
    hipblasLtHandle_t lt;
    stat = hipblasLtCreate(&lt);
    hipblasLtMatrixTransformDesc_t transformDesc;
    stat = hipblasLtMatrixTransformDescCreate(&transformDesc, HIP_R_8I);

    hipblasLtMatrixLayout_t d1_layout, d2_layout;
    stat = hipblasLtMatrixLayoutCreate(&d1_layout, HIP_R_8I, M, N, M);
    stat = hipblasLtMatrixLayoutCreate(&d2_layout, HIP_R_8I, M, N, 32 * 8 * 8);
    hipblasLtOrder_t target_order = CUBLASLT_ORDER_COL4_4R2_8C;
    stat = hipblasLtMatrixLayoutSetAttribute(d2_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &target_order, sizeof(target_order));
    stat = hipblasLtMatrixTransform(lt, transformDesc, &alpha, d1, d1_layout, &beta, nullptr, nullptr, d2, d2_layout, nullptr);
    hipError_t err = hipMemcpy(h2, d2, M * N * sizeof(int8_t), hipMemcpyDeviceToHost);
    err = hipDeviceSynchronize();
    printf("P1!\n");
    err = hipDeviceSynchronize();
    printf("P2!\n");
    
    hipblasLtMatrixLayoutDestroy(d1_layout);
    hipblasLtMatrixLayoutDestroy(d2_layout);
    hipblasLtMatrixTransformDescDestroy(transformDesc);
    hipblasLtDestroy(lt);
    free_memory(4, h1, h2, d1, d2);
    return 0;
}
