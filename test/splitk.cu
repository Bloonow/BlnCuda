#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include "../learn/helper.cu"

int main(int argc, char *argv[]) {
    int32_t Batch = 4;
    int64_t M = 64, N = 64, K = 16384;
    uint64_t workspace_bytes = 16 * 1024 * 1024;
    float alpha = 1.0, beta = 0;
    float *h_A = alloc_host_memory<float>(Batch * M * K);
    float *h_B = alloc_host_memory<float>(Batch * K * N);
    float *h_C = alloc_host_memory<float>(Batch * M * N);
    float *ret_C = alloc_host_memory<float>(Batch * M * N);
    float *d_A = alloc_cuda_memory<float>(Batch * M * K, h_A);
    float *d_B = alloc_cuda_memory<float>(Batch * K * N, h_B);
    float *d_C = alloc_cuda_memory<float>(Batch * M * N, h_C);
    void *workspace = alloc_cuda_memory<char>(workspace_bytes / sizeof(char), nullptr);

    hipblasLtHandle_t lt;
    hipblasStatus_t stat = hipblasLtCreate(&lt);

    // 设置矩阵布局
    hipblasLtOrder_t order = HIPBLASLT_ORDER_ROW;
    hipblasLtMatrixLayout_t A_layout, B_layout, C_layout;
    stat = hipblasLtMatrixLayoutCreate(&A_layout, HIP_R_32F, M, K, order == HIPBLASLT_ORDER_COL ? M : K);
    stat = hipblasLtMatrixLayoutCreate(&B_layout, HIP_R_32F, K, N, order == HIPBLASLT_ORDER_COL ? K : N);
    stat = hipblasLtMatrixLayoutCreate(&C_layout, HIP_R_32F, M, N, order == HIPBLASLT_ORDER_COL ? M : N);
    // 设置行主序存储或列主序存储
    stat = hipblasLtMatrixLayoutSetAttribute(A_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
    stat = hipblasLtMatrixLayoutSetAttribute(B_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
    stat = hipblasLtMatrixLayoutSetAttribute(C_layout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
    // 设置批量数目及跨步间距
    int64_t A_stride = M * K, B_stride = K * N, C_stride = M * N;
    stat = hipblasLtMatrixLayoutSetAttribute(A_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &A_stride, sizeof(A_stride));
    stat = hipblasLtMatrixLayoutSetAttribute(B_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &B_stride, sizeof(B_stride));
    stat = hipblasLtMatrixLayoutSetAttribute(C_layout, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &C_stride, sizeof(C_stride));
    stat = hipblasLtMatrixLayoutSetAttribute(A_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(Batch));
    stat = hipblasLtMatrixLayoutSetAttribute(B_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(Batch));
    stat = hipblasLtMatrixLayoutSetAttribute(C_layout, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &Batch, sizeof(Batch));

    // 设置矩阵乘法计算配置
    hipblasLtMatmulDesc_t matmul_desc;
    stat = hipblasLtMatmulDescCreate(&matmul_desc, HIPBLAS_COMPUTE_32F, HIP_R_32F);

    // has used hipblasLtMatrixLayoutSetAttribute() to set BatchCount and Strided
    const int requestAlgoCount = 4;
    int returnAlgoCount = 0;
    hipblasLtMatmulPreference_t preference;
    stat = hipblasLtMatmulPreferenceCreate(&preference);
    stat = hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspace_bytes, sizeof(workspace_bytes)
    );
    hipblasLtMatmulHeuristicResult_t heuristicResult[requestAlgoCount] = { 0 };
    stat = hipblasLtMatmulAlgoGetHeuristic(
        lt, matmul_desc, A_layout, B_layout, C_layout, C_layout, preference,
        requestAlgoCount, heuristicResult, &returnAlgoCount
    );
    hipblasLtMatmulAlgo_t algo = heuristicResult[0].algo;
    int32_t splitK = 16;  // set the split number for using SplitK algorithm
    stat = cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &splitK, sizeof(splitK));
    stat = hipblasLtMatmul(
        lt, matmul_desc, &alpha, d_A, A_layout, d_B, B_layout, &beta, d_C, C_layout, d_C, C_layout,
        &algo, workspace, workspace_bytes, nullptr
    );
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("[Error][%d], %s\n", stat, cublasLtGetStatusString(stat));
    }
    
    hipMemcpy(ret_C, d_C, sizeof(float) * Batch * M * N, hipMemcpyDeviceToHost);

    // 释放资源
    hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatmulDescDestroy(matmul_desc);
    hipblasLtMatrixLayoutDestroy(C_layout);
    hipblasLtMatrixLayoutDestroy(B_layout);
    hipblasLtMatrixLayoutDestroy(A_layout);
    hipblasLtDestroy(lt);
    free_memory(8, h_A, h_B, h_C, ret_C, d_A, d_B, d_C, workspace);
    return 0;
}